
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include <time.h> 


__global__ void onda2d(float *u, float *u_m1, float *u_p1,const float dt,const float dx,const float c,const int T) {
    int x = threadIdx.x ;
    int y = threadIdx.y ;
    int bx=blockIdx.x, bdimx=blockDim.x;
    int by=blockIdx.y, bdimy=blockDim.y;
    int Nx = blockDim.x *gridDim.x;
    int Ny = blockDim.y *gridDim.y;
    int idx= x + bx * bdimx;
    int idy= y + by * bdimy;

    __shared__ float temp[32*32];
    __shared__ float izquierda[32];
    __shared__ float derecha[32];
    __shared__ float arriba[32];
    __shared__ float abajo[32];



    if (idx>0 && idx<Nx-1 && idy>0 && idy<Ny-1){
        temp[y*bdimy+x]=u[idy*Nx+idx];
        for (int t=0;t<T;t++){
            if(x==blockDim.x-1){izquierda[y]=u[t*Ny*Nx+idy*Nx+idx+1];}
            else if(x==0){derecha[y]=u[t*Ny*Nx+idy*Nx+idx];}
            else if(y==blockDim.y-1){abajo[x]=u[t*Ny*Nx+(idy+1)*Nx+idx];}
            else if(y==0){arriba[x]=u[t*Ny*Nx+idy*Nx+idx];}
            __syncthreads();

            if(x==blockDim.x-1){
                u_p1[idy*bdimy+idx]=dt*dt/(dx*dx)*c*c*(izquierda[y]+temp[y*bdimy+x-1]+
                    temp[(y+1)*bdimy+x]+temp[(y-1)*bdimy+x]-4.0*temp[y*bdimy+x]) + 2.0*temp[y*bdimy+x]-u_m1[idy*bdimy+idx];
            }
            else if(x==0){
                u_p1[idy*bdimy+idx]=dt*dt/(dx*dx)*c*c*(temp[y*bdimy+x+1]+derecha[y]+
                    temp[(y+1)*bdimy+x]+temp[(y-1)*bdimy+x]-4.0*temp[y*bdimy+x]) + 2.0*temp[y*bdimy+x]-u_m1[idy*bdimy+idx];
            }
            else if(y==blockDim.y-1){
                u_p1[idy*bdimy+idx]=dt*dt/(dx*dx)*c*c*(temp[y*bdimy+x+1]+temp[y*bdimy+x-1]+
                    abajo[x]+temp[(y-1)*bdimy+x]-4.0*temp[y*bdimy+x]) + 2.0*temp[y*bdimy+x]-u_m1[idy*bdimy+idx];
            }
            
            else if(y==0){
                u_p1[idy*bdimy+idx]=dt*dt/(dx*dx)*c*c*(temp[y*bdimy+x+1]+temp[y*bdimy+x-1]+
                    temp[(y+1)*bdimy+x]+arriba[x]-4.0*temp[y*bdimy+x]) + 2.0*temp[y*bdimy+x]-u_m1[idy*bdimy+idx];
            }
            else{
                u_p1[idy*bdimy+idx]=dt*dt/(dx*dx)*c*c*(temp[y*bdimy+x+1]+temp[y*bdimy+x-1]+
                    temp[(y+1)*bdimy+x]+temp[(y-1)*bdimy+x]-4.0*temp[y*bdimy+x]) + 2.0*temp[y*bdimy+x]-u_m1[idy*bdimy+idx];
            }
            __syncthreads();

            u_m1[idy*Nx+idx]=temp[y*bdimy+x];
        
            temp[y*bdimy+x]=u_p1[idy*Nx+idx];
            u[t*Nx*Ny+idy*Ny+idx]=temp[y*bdimy+x];
            __syncthreads();
        }
    }
}

// llenar el array con los indices
void gauss(float *data,float t,float c,float dx,float dy,float y_0,float x_0,int Nx, int Ny) {
    float r;
    for (int y=1;y<Ny-1;y++){
        for (int x=1;x<Nx-1;x++){
            r=sqrt((x*dx-x_0)*(x*dx-x_0)+(y*dy-y_0)*(y*dy-y_0));
		    data[y*Ny+x] = 2*exp(-(r-c*t)*(r-c*t)/4);
        }
    }
}   
void guardar_salida(float *data,int Nx, int Ny,int T) {
    FILE *fp = fopen("onda_2d_cuda.dat", "w");

    for (int t=0;t<T;t++){
        for (int y=0;y<Ny;y++){
            for (int x=0;x<Nx;x++){
                fprintf(fp, "%g\n", data[t*Nx*Ny+y*Ny+x]);
            }
        }
    }
}
int main(int argc, char *argv[]){
	float  *u, *u_m1, *u_p1,*d_u, *d_u_m1, *d_u_p1;

    int Nx=atoi(argv[1]);
    int Ny=atoi(argv[2]);
    int T=atoi(argv[3]);
    float c=atoi(argv[4]);

	int size = Nx*Ny*sizeof(float);

    float dx=1.0;
    float dy=1.0;
    float dt=1.0/T;

    double time_spent = 0.0;
    clock_t begin = clock();

	u = (float *)malloc(size*T); gauss(u,dt,c,dx,dy, Nx/2,Ny/2,Nx,Ny);
	u_m1 = (float *)malloc(size); gauss(u_m1,0,c,dx,dy, Nx/2, Ny/2,Nx,Ny);
	u_p1 = (float *)malloc(size);

    


    // Asignar memoria al lado del device (GPU)
    hipMalloc((void **)&d_u, size*T);
    hipMalloc((void **)&d_u_m1, size);
    hipMalloc((void **)&d_u_p1, size);

    // Copiar al device
    hipMemcpy(d_u, u, size*T, hipMemcpyHostToDevice);
    hipMemcpy(d_u_m1, u_m1, size, hipMemcpyHostToDevice);

    // Invocar kernel con N bloques de 1 thread cada uno
    dim3 bloque (Nx/2,Ny/2);
    dim3 grid (2,2);
    onda2d<<<grid,bloque>>>(d_u,d_u_m1,d_u_p1,dt,dx,c,T);
    
    hipError_t err = hipGetLastError();
    printf("Error: %s\n",hipGetErrorString(err));

    // Copiar resultado al host
    hipMemcpy(u, d_u, size*T, hipMemcpyDeviceToHost);
    guardar_salida(u,Nx,Ny,T);
    clock_t end = clock();
 

    time_spent += (double)(end - begin) / CLOCKS_PER_SEC;
 
    printf("todo %f seconds\n", time_spent);

	free(u); free(u_m1); free(u_p1);
    hipFree(d_u); hipFree(d_u_m1); hipFree(d_u_p1);

	return 0;
}

