#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include <time.h> 


__global__ void onda2d(float *u, float *u_m1, float *u_p1,float dt,float dx,float c,int T,int Nx, int Ny,int t) {
    int x = threadIdx.x + blockDim.x*blockIdx.x;
    int y = threadIdx.y + blockDim.y*blockIdx.y;

    if (x>0 && x<Nx-1 && y>0 && y<Ny-1){

            u_p1[y*Ny+x]=dt*dt/(dx*dx)*c*c*(u[t*Nx*Ny+y*Ny+x+1]+u[t*Nx*Ny+y*Ny+x-1]+
                u[t*Nx*Ny+(y+1)*Ny+x]+u[t*Nx*Ny+(y-1)*Ny+x]-4.0*u[t*Nx*Ny+y*Ny+x]) + 2.0*u[t*Nx*Ny+y*Ny+x] - u_m1[y*Ny+x];
            
		u_m1[y*Ny+x]=u[t*Nx*Ny+y*Ny+x];

            u[(t+1)*Nx*Ny+y*Ny+x]=u_p1[y*Ny+x];
    

    }
     __syncthreads();


}




// llenar el array con los indices
void gauss(float *data,float t,float c,float dx,float dy,float y_0,float x_0,int Nx, int Ny) {
    float r;
    for (int y=1;y<Ny-1;y++){
        for (int x=1;x<Nx-1;x++){
            r=sqrt((x*dx-x_0)*(x*dx-x_0)+(y*dy-y_0)*(y*dy-y_0));
		    data[y*Ny+x] = 2*exp(-(r-c*t)*(r-c*t)/4);
        }
    }
}   
void guardar_salida(float *data,int Nx, int Ny,int T) {

    FILE *fp = fopen("onda_2d_cuda.dat", "w");
	
	fwrite(&(data[0]),sizeof(float),Nx*Ny*T,fp);
	fclose(fp);
}

int main(int argc, char *argv[]){
	float  *u_s1, *u_m1_s1, *u_p1_s1,*d_u_s1, *d_u_m1_s1, *d_u_p1_s1;
        float  *u_s2, *u_m1_s2, *u_p1_s2,*d_u_s2, *d_u_m1_s2, *d_u_p1_s2;


    int Nx=atoi(argv[1]);
    int Ny=atoi(argv[2]);
    int T=atoi(argv[3]);
    float c=atoi(argv[4]);

	int size = Nx*Ny*sizeof(float);

    float dx=1.0;
    float dy=1.0;
    float dt=1.0/T;

    double time_spent = 0.0;
    clock_t begin = clock();

	u_s1 = (float *)malloc(size*T); gauss(u_s1,dt,c,dx,dy, 3*Nx/4,3*Ny/4,Nx,Ny);
	u_m1_s1 = (float *)malloc(size); gauss(u_m1_s1,0,c,dx,dy, 3*Nx/4, 3*Ny/4,Nx,Ny);
	u_p1_s1 = (float *)malloc(size);

        u_s2 = (float *)malloc(size*T); gauss(u_s2,dt,c,dx,dy, Nx/4,Ny/4,Nx,Ny);
        u_m1_s2 = (float *)malloc(size); gauss(u_m1_s2,0,c,dx,dy, Nx/4, Ny/4,Nx,Ny);
        u_p1_s2 = (float *)malloc(size);




    // Asignar memoria al lado del device (GPU)
    hipMalloc((void **)&d_u_s1, size*T);
    hipMalloc((void **)&d_u_m1_s1, size);
    hipMalloc((void **)&d_u_p1_s1, size);


    hipMalloc((void **)&d_u_s2, size*T);
    hipMalloc((void **)&d_u_m1_s2, size);
    hipMalloc((void **)&d_u_p1_s2, size);

hipStream_t stream1, stream2;
hipStreamCreate ( &stream1) ;
hipStreamCreate ( &stream2) ;

   
    // Copiar al device
    hipMemcpyAsync(d_u_s1, u_s1, size*T, hipMemcpyHostToDevice,stream1);
    hipMemcpyAsync(d_u_m1_s1, u_m1_s1, size, hipMemcpyHostToDevice,stream1);

    hipMemcpyAsync(d_u_s2, u_s2, size*T, hipMemcpyHostToDevice,stream2);
    hipMemcpyAsync(d_u_m1_s2, u_m1_s2, size, hipMemcpyHostToDevice,stream2);


    // Invocar kernel con N bloques de 1 thread cada uno
    dim3 bloque (32,32);
    dim3 grid(2,2);

    printf("%d %d\n",(int)ceil(Nx/32),(int)ceil(Ny/32));
    

    for (int t=0;t<T;t++){ 
    	onda2d<<<grid,bloque,0,stream1>>>(d_u_s1,d_u_m1_s1,d_u_p1_s1,dt,dx,c,T,Nx,Ny,t);
        onda2d<<<grid,bloque,0,stream2>>>(d_u_s2,d_u_m1_s2,d_u_p1_s2,dt,dx,c,T,Nx,Ny,t);

   }
    hipError_t err = hipGetLastError();
    printf("Error: %s\n",hipGetErrorString(err));

    // Copiar resultado al host
       hipMemcpyAsync(u_s2, d_u_s2, size*T, hipMemcpyDeviceToHost,stream1);
       hipMemcpyAsync(u_s1, d_u_s1, size*T, hipMemcpyDeviceToHost,stream2);
	
	float *phi_final=u_s1+u_s2;

	 guardar_salida(phi_final,Nx,Ny,T);
    clock_t end = clock();
 

    time_spent += (double)(end - begin) / CLOCKS_PER_SEC;
 
    printf("todo %f seconds\n", time_spent);

	free(u_s1); free(u_m1_s1); free(u_p1_s1);
    hipFree(d_u_s1); hipFree(d_u_m1_s1); hipFree(d_u_p1_s1);

        free(u_s2); free(u_m1_s2); free(u_p1_s2);
    hipFree(d_u_s2); hipFree(d_u_m1_s2); hipFree(d_u_p1_s2);

	return 0;
}
