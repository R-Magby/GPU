
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include <time.h> 


__global__ void onda2d(float *u, float *u_m1, float *u_p1,const float dt,const float dx,const float c,const int T) {
    int x = threadIdx.x ;
    int y = threadIdx.y ;
    int Nx = blockDim.x ;
    int Ny = blockDim.y ;
    __shared__ float temp[32*32];

    if (x>0 && x<Nx-1 && y>0 && y<Ny-1){
        temp[y*Ny+x]=u[y*Ny+x];

        for (int t=0;t<T;t++){
            
            u_p1[y*Ny+x]=dt*dt/(dx*dx)*c*c*(temp[y*Ny+x+1]+temp[y*Ny+x-1]+
                temp[(y+1)*Ny+x]+temp[(y-1)*Ny+x]-4.0*temp[y*Ny+x]) + 2.0*temp[y*Ny+x]-u_m1[y*Ny+x];
            __syncthreads();

            u_m1[y*Ny+x]=temp[y*Ny+x];
        
            temp[y*Ny+x]=u_p1[y*Ny+x];
            u[t*Nx*Ny+y*Ny+x]=temp[y*Ny+x];
            __syncthreads();

        }
    }
}


// llenar el array con los indices
void gauss(float *data,float t,float c,float dx,float dy,float y_0,float x_0,int Nx, int Ny) {
    float r;
    for (int y=1;y<Ny-1;y++){
        for (int x=1;x<Nx-1;x++){
            r=sqrt((x*dx-x_0)*(x*dx-x_0)+(y*dy-y_0)*(y*dy-y_0));
		    data[y*Ny+x] = 2*exp(-(r-c*t)*(r-c*t)/4);
        }
    }
}   
void guardar_salida(float *data,int Nx, int Ny,int T) {
    FILE *fp = fopen("onda_2d_cuda.dat", "w");

    for (int t=0;t<T;t++){
        for (int y=0;y<Ny;y++){
            for (int x=0;x<Nx;x++){
                fprintf(fp, "%g\n", data[t*Nx*Ny+y*Ny+x]);
            }
        }
    }
}
int main(int argc, char *argv[]){
	float  *u, *u_m1, *u_p1,*d_u, *d_u_m1, *d_u_p1;

    int Nx=atoi(argv[1]);
    int Ny=atoi(argv[2]);
    int T=atoi(argv[3]);
    float c=atoi(argv[4]);

	int size = Nx*Ny*sizeof(float);

    float dx=1.0;
    float dy=1.0;
    float dt=1.0/T;

    double time_spent = 0.0;
    clock_t begin = clock();

	u = (float *)malloc(size*T); gauss(u,dt,c,dx,dy, Nx/2,Ny/2,Nx,Ny);
	u_m1 = (float *)malloc(size); gauss(u_m1,0,c,dx,dy, Nx/2, Ny/2,Nx,Ny);
	u_p1 = (float *)malloc(size);

    


    // Asignar memoria al lado del device (GPU)
    hipMalloc((void **)&d_u, size*T);
    hipMalloc((void **)&d_u_m1, size);
    hipMalloc((void **)&d_u_p1, size);

    // Copiar al device
    hipMemcpy(d_u, u, size*T, hipMemcpyHostToDevice);
    hipMemcpy(d_u_m1, u_m1, size, hipMemcpyHostToDevice);

    // Invocar kernel con N bloques de 1 thread cada uno
    dim3 bloque (Nx,Ny);
    onda2d<<<1,bloque>>>(d_u,d_u_m1,d_u_p1,dt,dx,c,T);
    
    hipError_t err = hipGetLastError();
    printf("Error: %s\n",hipGetErrorString(err));

    // Copiar resultado al host
    hipMemcpy(u, d_u, size*T, hipMemcpyDeviceToHost);
    guardar_salida(u,Nx,Ny,T);
    clock_t end = clock();
 

    time_spent += (double)(end - begin) / CLOCKS_PER_SEC;
 
    printf("todo %f seconds\n", time_spent);

	free(u); free(u_m1); free(u_p1);
    hipFree(d_u); hipFree(d_u_m1); hipFree(d_u_p1);

	return 0;
}

