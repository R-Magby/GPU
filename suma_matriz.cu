#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include <time.h> 

__global__ void suma_matriz(int *A,int *B,int *C,int Nx){
	int idx=threadIdx.x + blockDim.x*blockIdx.x;
	int idy=threadIdx.y + blockDim.y*blockIdx.y;
	

	C[idy*Nx+idx] = A[idy*Nx+idx]+B[idy*Nx+idx];

}

void rellenar(int *A,int Nx,int Ny){
	for(int i=0;i<Nx;i++){
		for(int j=0;j<Ny;j++){
			A[j*Nx+i]=j*Nx+i;
		}
	}
}

main(int argc,char *argv[]){

	int *A,*B,*C,*cuA,*cuB,*cuC;
        int *D,*E,*F,*cuD,*cuE,*cuF;


	int Nx=atoi(argv[1]);
        int Ny=atoi(argv[2]);

	int size=Nx*Ny*sizeof(int);

	A=(int*)malloc(size);
	B=(int*)malloc(size);
	C=(int*)malloc(size);
	rellenar(A,Nx,Ny);
	rellenar(B,Nx,Ny);
	

        hipMalloc((void **)&cuA, size);
        hipMalloc((void **)&cuB, size);
        hipMalloc((void **)&cuC, size);


        D=(int*)malloc(size);
        E=(int*)malloc(size);
        F=(int*)malloc(size);
        rellenar(D,Nx,Ny);
        rellenar(E,Nx,Ny);


        hipMalloc((void **)&cuD, size);
        hipMalloc((void **)&cuE, size);
        hipMalloc((void **)&cuF, size);


	hipMemcpy(cuA,A,size,hipMemcpyHostToDevice);
        hipMemcpy(cuB,B,size,hipMemcpyHostToDevice);

        hipMemcpy(cuD,D,size,hipMemcpyHostToDevice);
        hipMemcpy(cuE,E,size,hipMemcpyHostToDevice);


	dim3 grid(1,1);
	dim3 bloque(32,32);
	hipStream_t stream1, stream2 ;
	
	hipStreamCreate ( &stream1) ;
	hipStreamCreate ( &stream2) ;

//	hipMemcpyAsync ( cuA, A, size, hipMemcpyHostToDevice, stream1 ) ;


	suma_matriz<<<grid,bloque,0,stream1>>>(cuA,cuB,cuC,Nx);
	suma_matriz<<<grid,bloque,0,stream2>>>(cuD,cuE,cuF,Nx);


    hipError_t err = hipGetLastError();
    printf("Error: %s\n",hipGetErrorString(err));

    // Copiar resultado al host
    hipMemcpy(C, cuC, size, hipMemcpyDeviceToHost);
    hipMemcpy(F, cuF, size, hipMemcpyDeviceToHost);

	for(int i=0;i<Nx;i++){
                for(int j=0;j<Ny;j++){
                        printf("%d |",C[j*Nx+i]);
                        printf("%d \n",F[j*Nx+i]);

                }
        }

	free(A); free(B); free(C);
    hipFree(cuA); hipFree(cuB); hipFree(cuC);

	return 0;
}


